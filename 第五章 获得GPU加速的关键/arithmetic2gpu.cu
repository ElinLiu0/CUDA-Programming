#include "hip/hip_runtime.h"
#include "error.cuh"
#include <math.h>
#include <stdio.h>
// 定义宏
#ifdef USE_DP
    typedef double real;
#else
    typedef float real;
#endif
// 定义循环次数
const int NUM_REPEATS = 10;
// 定义x0为整数数据
const real x0 = 100.0;
// 初始化arthmetic函数，参数为x指针，x0和N
void __global__ arithmetic(real *x, const real x0, const int N);

int main(int argc,char **argv)
{
    if(argc != 2)
    {
        printf("Usage: %s <N>\n", argv[0]);
        exit(1);
    };
    // 初始化数组长度
    // atoi函数是C++中将字符串直接转换为整数数据类型的函数
    const int N = atoi(argv[1]);
    // 初始化数组内存大小
    const int M = sizeof(real) * N;
    // 初始化block_size和grid_size
    const int block_size = 128;
    const int grid_size = (N + block_size - 1) / block_size;
    real *h_x = (real*) malloc(M);
    // 初始化GPU内存指针
    real *d_x;
    // 分配GPU内存
    CHECK(hipMalloc((void **)&d_x, M));
    // 初始化时间变量
    float t_sum = 0;
    float t2_sum = 0;
    for (int repeat = 0; repeat <= NUM_REPEATS; ++repeat)
    {
        // 为数组赋值，期间GPU将执行10 * 10000，也就是十万次该操作
        for (int n = 0; n < N; ++n)
        {
            h_x[n] = 0.0;
        }
        // 初始化hipEvent_t 对象用于表示开始和结束的cudaEvent事件
        hipEvent_t start, stop;
        // 创建hipEvent_t 对象
        CHECK(hipEventCreate(&start));
        CHECK(hipEventCreate(&stop));
        // 记录开始值
        CHECK(hipEventRecord(start));
        hipEventQuery(start);
        // 调用核函数
        arithmetic<<<grid_size, block_size>>>(d_x, x0, N);
        // 记录结束事件
        CHECK(hipEventRecord(stop));
        // 同步结束事件
        CHECK(hipEventSynchronize(stop));
        // 计算时间
        float elapsed_time;
        CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
        printf("Time = %g ms.\n", elapsed_time);
        // 防止内核初始化造成的性能准确度降低，因此记录第二次往后的时间
        if (repeat > 0)
        {
            t_sum += elapsed_time;
            t2_sum += elapsed_time * elapsed_time;
        }
        // 销毁CUDA事件
        CHECK(hipEventDestroy(start));
        CHECK(hipEventDestroy(stop));
    }
    // 计算平均时间
    const float t_ave = t_sum / NUM_REPEATS;
    const float t_err = sqrt(t2_sum / NUM_REPEATS - t_ave * t_ave);
    printf("Time = %g +- %g ms.\n", t_ave, t_err);
    // 释放内存
    free(h_x);
    CHECK(hipFree(d_x));
    return 0;
}

void __global__ arithmetic(real *d_x,const real x0,const int N)
{
    const int n = blockDim.x * blockIdx.x + threadIdx.x;
    if (n < N)
    {
        real x_tmp = d_x[n];
        while(sqrt(x_tmp) < x0)
        {
            ++x_tmp;
        }
        d_x[n] = x_tmp;
    }
}
