
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(void){
    printf("Hello from GPU!\n");
}

int main(void){
    /*
        此时cuda内核将调用总共8个线程来运行程序，
        其中2代表gridNum，4代表BlockNum,
        此时函数也将被调用2*4=8次
    */
    hello_from_gpu<<<2,4>>>();
    /*
        注意这里blockId不一定是0还是1，
        因此这也说明CUDA在执行程序的高度随机化
    */
    hipDeviceSynchronize();
    return 0;
}